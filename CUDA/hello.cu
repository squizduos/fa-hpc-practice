
#include "hip/hip_runtime.h"

#include <stdio.h>


// __device__ - GPU
// __global__ - GPU 
// __host__   - CPU
__global__ void add( int a, int b, int *c)
{
  *c = a + b;
}

int main()
{
  int c;
  int *dev_c;

  hipMalloc( (void**)&dev_c, sizeof(int));

  add<<<1,1>>> (1, 2, dev_c);

  hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

  printf("%i\n", c);

  hipFree(dev_c);
  return 0;
}
